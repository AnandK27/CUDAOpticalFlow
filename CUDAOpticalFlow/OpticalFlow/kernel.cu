#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <opencv2/opencv.hpp>

#include <stdio.h>
#include <iostream>

#define BLOCK_SIZE 32
#define WIN_SIZE 3
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// kernel to get the Ix, Iy and It of two images
__global__ void getDerivatives(int* Ix, int* Iy, int* It, int* I1, int* I2, int width, int height)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int idx = i + j * width;
	if (i > 0 && i < width - 1 && j > 0 && j < height - 1)
	{
		Ix[idx] = (I1[idx + 1] - I1[idx - 1] + I2[idx + 1] - I2[idx - 1]) / 4;
		Iy[idx] = (I1[idx + width] - I1[idx - width] + I2[idx + width] - I2[idx - width]) / 4;
		It[idx] = (I2[idx] - I1[idx]);
	}
}

//kernel to get Ix, Iy and It of two images using shared memory
__global__ void getDerivatives_2(int* Ix, int* Iy, int* It, int* I1, int* I2, int width, int height)
{	
	__shared__ int I1_shared[BLOCK_SIZE + 2][BLOCK_SIZE + 2];
	__shared__ int I2_shared[BLOCK_SIZE + 2][BLOCK_SIZE + 2];

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int idx = i + j * width;

	I1_shared[threadIdx.y + 1][threadIdx.x + 1] = I1[idx];
	I2_shared[threadIdx.y + 1][threadIdx.x + 1] = I2[idx];


	if (threadIdx.x == 0 && i > 0)
	{
		I1_shared[threadIdx.y + 1][threadIdx.x] = I1[idx - 1];
		I2_shared[threadIdx.y + 1][threadIdx.x] = I2[idx - 1];
	}

	if (threadIdx.x == blockDim.x - 1 && i < width - 1)
	{
		I1_shared[threadIdx.y + 1][threadIdx.x + 2] = I1[idx + 1];
		I2_shared[threadIdx.y + 1][threadIdx.x + 2] = I2[idx + 1];
	}

	if (threadIdx.y == 0 && j > 0)
	{
		I1_shared[threadIdx.y][threadIdx.x + 1] = I1[idx - width];
		I2_shared[threadIdx.y][threadIdx.x + 1] = I2[idx - width];
	}

	if (threadIdx.y == blockDim.y - 1 && j < height - 1)
	{
		I1_shared[threadIdx.y + 2][threadIdx.x + 1] = I1[idx + width];
		I2_shared[threadIdx.y + 2][threadIdx.x + 1] = I2[idx + width];
	}

	// Wait for all threads to finish copying
	__syncthreads();

	if (i > 0 && i < width - 1 && j > 0 && j < height - 1) {
		Ix[idx] = (I1_shared[threadIdx.y + 1][threadIdx.x + 2] - I1_shared[threadIdx.y + 1][threadIdx.x] + I2_shared[threadIdx.y + 1][threadIdx.x + 2] - I2_shared[threadIdx.y + 1][threadIdx.x]) / 4;
		Iy[idx] = (I1_shared[threadIdx.y + 2][threadIdx.x + 1] - I1_shared[threadIdx.y][threadIdx.x + 1] + I2_shared[threadIdx.y + 2][threadIdx.x + 1] - I2_shared[threadIdx.y][threadIdx.x + 1]) / 4;
		It[idx] = (I2_shared[threadIdx.y + 1][threadIdx.x + 1] - I1_shared[threadIdx.y + 1][threadIdx.x + 1]);
	}
}

__global__ void computeOpticalFlow_GPU(int* Ix, int* Iy, int* It, int width, int height, int stride, int* u, int* v)
{

	__shared__ int Ix_shared[BLOCK_SIZE + WIN_SIZE - 1][BLOCK_SIZE + WIN_SIZE - 1];
	__shared__ int Iy_shared[BLOCK_SIZE + WIN_SIZE - 1][BLOCK_SIZE + WIN_SIZE - 1];
	__shared__ int It_shared[BLOCK_SIZE + WIN_SIZE - 1][BLOCK_SIZE + WIN_SIZE - 1];

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	int idx = y * stride + x;

	Ix_shared[threadIdx.y + WIN_SIZE / 2][threadIdx.x + WIN_SIZE / 2] = Ix[idx];
	Iy_shared[threadIdx.y + WIN_SIZE / 2][threadIdx.x + WIN_SIZE / 2] = Iy[idx];
	It_shared[threadIdx.y + WIN_SIZE / 2][threadIdx.x + WIN_SIZE / 2] = It[idx];

	if (threadIdx.x < WIN_SIZE / 2 && x > 0)
	{
		Ix_shared[threadIdx.y + WIN_SIZE / 2][threadIdx.x] = Ix[idx - WIN_SIZE / 2];
		Iy_shared[threadIdx.y + WIN_SIZE / 2][threadIdx.x] = Iy[idx - WIN_SIZE / 2];
		It_shared[threadIdx.y + WIN_SIZE / 2][threadIdx.x] = It[idx - WIN_SIZE / 2];
	}

	if (threadIdx.x >= blockDim.x - WIN_SIZE / 2 && x < width - 1)
	{
		Ix_shared[threadIdx.y + WIN_SIZE / 2][threadIdx.x + WIN_SIZE - 1] = Ix[idx + WIN_SIZE / 2];
		Iy_shared[threadIdx.y + WIN_SIZE / 2][threadIdx.x + WIN_SIZE - 1] = Iy[idx + WIN_SIZE / 2];
		It_shared[threadIdx.y + WIN_SIZE / 2][threadIdx.x + WIN_SIZE - 1] = It[idx + WIN_SIZE / 2];
	}

	if (threadIdx.y < WIN_SIZE / 2 && y > 0)
	{
		Ix_shared[threadIdx.y][threadIdx.x + WIN_SIZE / 2] = Ix[idx - WIN_SIZE / 2 * stride];
		Iy_shared[threadIdx.y][threadIdx.x + WIN_SIZE / 2] = Iy[idx - WIN_SIZE / 2 * stride];
		It_shared[threadIdx.y][threadIdx.x + WIN_SIZE / 2] = It[idx - WIN_SIZE / 2 * stride];
	}

	if (threadIdx.y >= blockDim.y - WIN_SIZE / 2 && y < height - 1)
	{
		Ix_shared[threadIdx.y + WIN_SIZE - 1][threadIdx.x + WIN_SIZE / 2] = Ix[idx + WIN_SIZE / 2 * stride];
		Iy_shared[threadIdx.y + WIN_SIZE - 1][threadIdx.x + WIN_SIZE / 2] = Iy[idx + WIN_SIZE / 2 * stride];
		It_shared[threadIdx.y + WIN_SIZE - 1][threadIdx.x + WIN_SIZE / 2] = It[idx + WIN_SIZE / 2 * stride];
	}

	__syncthreads();
	if (x < WIN_SIZE / 2 || x >= width - WIN_SIZE / 2 || y < WIN_SIZE / 2 || y >= height - WIN_SIZE / 2)
	{
		u[y * stride + x] = 0;
		v[y * stride + x] = 0;
		return;
	}

	float sumIx2 = 0, sumIy2 = 0, sumIxIy = 0, sumIxIt = 0, sumIyIt = 0;
	// Compute the sums
	for (int wy = -WIN_SIZE / 2; wy <= WIN_SIZE / 2; wy++)
	{
		for (int wx = -WIN_SIZE / 2; wx <= WIN_SIZE / 2; wx++)
		{
			sumIx2 += Ix_shared[threadIdx.y + wy + WIN_SIZE / 2][threadIdx.x + wx + WIN_SIZE / 2] * Ix_shared[threadIdx.y + wy + WIN_SIZE / 2][threadIdx.x + wx + WIN_SIZE / 2];
			sumIy2 += Iy_shared[threadIdx.y + wy + WIN_SIZE / 2][threadIdx.x + wx + WIN_SIZE / 2] * Iy_shared[threadIdx.y + wy + WIN_SIZE / 2][threadIdx.x + wx + WIN_SIZE / 2];
			sumIxIy += Ix_shared[threadIdx.y + wy + WIN_SIZE / 2][threadIdx.x + wx + WIN_SIZE / 2] * Iy_shared[threadIdx.y + wy + WIN_SIZE / 2][threadIdx.x + wx + WIN_SIZE / 2];
			sumIxIt += Ix_shared[threadIdx.y + wy + WIN_SIZE / 2][threadIdx.x + wx + WIN_SIZE / 2] * It_shared[threadIdx.y + wy + WIN_SIZE / 2][threadIdx.x + wx + WIN_SIZE / 2];
			sumIyIt += Iy_shared[threadIdx.y + wy + WIN_SIZE / 2][threadIdx.x + wx + WIN_SIZE / 2] * It_shared[threadIdx.y + wy + WIN_SIZE / 2][threadIdx.x + wx + WIN_SIZE / 2];
		}
	}

	float det = sumIx2 * sumIy2 - sumIxIy * sumIxIy;
	if (fabs(det) > 1e-6) {
		u[y * stride + x] = (sumIy2 * -sumIxIt - sumIxIy * -sumIyIt) / det;
		v[y * stride + x] = (sumIx2 * -sumIyIt - sumIxIy * -sumIxIt) / det;
	}
	else {
		u[y * stride + x] = 0;
		v[y * stride + x] = 0;
	}
}

// Compute image gradients using Sobel operator
void computeGradients(int* gray, int width, int height, int stride, int* Ix, int* Iy, int* It, int* prevGray) {
    for (int y = 1; y < height - 1; y++) {
        for (int x = 1; x < width - 1; x++) {
            int idx = y * stride + x;

            Ix[idx] = (gray[idx + 1] - gray[idx - 1] + prevGray[idx + 1] - prevGray[idx - 1]) / 4.0f;
            Iy[idx] = (gray[idx + stride] - gray[idx - stride] + prevGray[idx + stride] - prevGray[idx - stride]) / 4.0f;
            It[idx] = (float)(gray[idx] - prevGray[idx]);
        }
    }
}


// Lucas-Kanade method
void computeOpticalFlow(int* Ix, int* Iy, int* It, int width, int height, int stride, int* u, int* v) {
	for (int y = WIN_SIZE / 2; y < height - WIN_SIZE / 2; y++) {
		for (int x = WIN_SIZE / 2; x < width - WIN_SIZE / 2; x++) {
			float sumIx2 = 0, sumIy2 = 0, sumIxIy = 0, sumIxIt = 0, sumIyIt = 0;

			for (int wy = -WIN_SIZE / 2; wy <= WIN_SIZE / 2; wy++) {
				for (int wx = -WIN_SIZE / 2; wx <= WIN_SIZE / 2; wx++) {
					int idx = (y + wy) * stride + (x + wx);
					sumIx2 += Ix[idx] * Ix[idx];
					sumIy2 += Iy[idx] * Iy[idx];
					sumIxIy += Ix[idx] * Iy[idx];
					sumIxIt += Ix[idx] * It[idx];
					sumIyIt += Iy[idx] * It[idx];
				}
			}

			float det = sumIx2 * sumIy2 - sumIxIy * sumIxIy;
			if (fabs(det) > 1e-6) {
				u[y * stride + x] = (sumIy2 * -sumIxIt - sumIxIy * -sumIyIt) / det;
				v[y * stride + x] = (sumIx2 * -sumIyIt - sumIxIy * -sumIxIt) / det;
			}
			else {
				u[y * stride + x] = 0;
				v[y * stride + x] = 0;
			}
		}
	}
}

int main()
{
    // Open video
    cv::VideoCapture cap("C:/Users/anand/Downloads/vehicles.mp4");
    if (!cap.isOpened()) {
        std::printf("Error: Unable to open video file.\n");
        return -1;
    }

    int width = 512;
    int height = 512;
    std::cout << width << " " << height << std::endl;
    int stride = width;

	// Allocate gpu host memory for the images
	int* I1 = new int[width * height];
	int* I2 = new int[width * height];
	int* Ix = new int[width * height];
	int* Iy = new int[width * height];
	int* It = new int[width * height];
	int* u = new int[width * height];
	int* v = new int[width * height];

	// initialize to zero
	for (int i = 0; i < width * height; i++) {
		I1[i] = 0;
		I2[i] = 0;
		Ix[i] = 0;
		Iy[i] = 0;
		It[i] = 0;
		u[i] = 0;
		v[i] = 0;
	}

	// Read the first frame
	cv::Mat frame;
	cap >> frame;
	cv::cvtColor(frame, frame, cv::COLOR_BGR2GRAY);
	cv::resize(frame, frame, cv::Size(width, height));
    for (int i = 0; i < width; i++) {
		for (int j = 0; j < height; j++) {
			I1[i + j * stride] = frame.at<uchar>(j, i);
		}
    }

	// Read the second frame
	cap >> frame;
	cv::cvtColor(frame, frame, cv::COLOR_BGR2GRAY);
	cv::resize(frame, frame, cv::Size(width, height));
	for (int i = 0; i < width; i++) {
		for (int j = 0; j < height; j++) {
			I2[i + j * stride] = frame.at<uchar>(j, i);
		}
	}

	// Allocate gpu memory for the images
	int* d_I1, * d_I2, * d_Ix, * d_Iy, * d_It, * d_u, * d_v;
	hipMalloc(&d_I1, width * height * sizeof(int));
	hipMalloc(&d_I2, width * height * sizeof(int));
	hipMalloc(&d_Ix, width * height * sizeof(int));
	hipMalloc(&d_Iy, width * height * sizeof(int));
	hipMalloc(&d_It, width * height * sizeof(int));
	hipMalloc(&d_u, width * height * sizeof(int));
	hipMalloc(&d_v, width * height * sizeof(int));

	// Copy the images to the gpu memory
	hipMemcpy(d_I1, I1, width * height * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_I2, I2, width * height * sizeof(int), hipMemcpyHostToDevice);
    
	// Launch the kernel to get the derivatives
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 numBlocks(width / threadsPerBlock.x, height / threadsPerBlock.y);
	getDerivatives_2<<<numBlocks, threadsPerBlock>>> (d_Ix, d_Iy, d_It, d_I1, d_I2, width, height);

	// Launch the kernel to compute the optical flow
	computeOpticalFlow_GPU << <numBlocks, threadsPerBlock >> > (d_Ix, d_Iy, d_It, width, height, stride, d_u, d_v);

	// Copy the derivatives back to the host memory
	hipMemcpy(Ix, d_Ix, width * height * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(Iy, d_Iy, width * height * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(It, d_It, width * height * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(u, d_u, width * height * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(v, d_v, width * height * sizeof(int), hipMemcpyDeviceToHost);

	// Free the gpu memory
	hipFree(d_I1);
	hipFree(d_I2);
	hipFree(d_Ix);
	hipFree(d_Iy);
	hipFree(d_It);
	hipFree(d_u);
	hipFree(d_v);

	//cpu code
	int* Ix_cpu = new int[width * height];
	int* Iy_cpu = new int[width * height];
	int* It_cpu = new int[width * height];
	int* u_cpu = new int[width * height];
	int* v_cpu = new int[width * height];

	//initialize to zero
	for (int i = 0; i < width * height; i++) {
		Ix_cpu[i] = 0;
		Iy_cpu[i] = 0;
		It_cpu[i] = 0;
		u_cpu[i] = 0;
		v_cpu[i] = 0;
	}

	computeGradients(I2, width, height, stride, Ix_cpu, Iy_cpu, It_cpu, I1);
	computeOpticalFlow(Ix_cpu, Iy_cpu, It_cpu, width, height, stride, u_cpu, v_cpu);


	// Compare the results
	float error = 0;
	for (int i = 0; i < width * height; i++) {
		if (abs(Ix[i] - Ix_cpu[i]) > 0) {
			std::cout << "Ix:" << i << " " << Ix[i] << " " << Ix_cpu[i] << std::endl;
		}
		if (abs(Iy[i] - Iy_cpu[i]) > 0) {
			std::cout << "Iy:" << i << " " << Ix[i] << " " << Ix_cpu[i] << std::endl;
		}
		if (abs(It[i] - It_cpu[i]) > 0) {
			std::cout << "It:" << i << " " << It[i] << " " << It_cpu[i] << std::endl;
		}
		if (abs(v[i] - v_cpu[i]) > 0) {
			std::cout << "u:" << i << " " << u[i] << " " << u_cpu[i] << std::endl;
			std::cout << "v:" << i << " " << v[i] << " " << v_cpu[i] << std::endl;
		}
		error += abs(Ix[i] - Ix_cpu[i]) + abs(Iy[i] - Iy_cpu[i]) + abs(It[i] - It_cpu[i]) + abs(u[i] - u_cpu[i]) + abs(v[i] - v_cpu[i]);
	}
	std::cout << "Error: " << error << std::endl;

	// Free the host memory
	delete[] I1;
	delete[] I2;
	delete[] Ix;
	delete[] Iy;
	delete[] It;
	delete[] u;
	delete[] v;
	std::cout << "Done" << std::endl;
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
