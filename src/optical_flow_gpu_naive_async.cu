#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <>
//#include <hip/device_functions.h>
using namespace std;

#define WIN_SIZE 5  // Window size for Lucas-Kanade method
#define BLOCK_SIZE 32

#define DISPLAY_STREAMS false
#define CPU false
#define DEBUG false

#define N 4

#define CHECK(call) \
{ \
    hipError_t error = call;                   \
    if (error != hipSuccess) \
    { \
        printf("Error in File: %s, Line: %d\n", __FILE__, __LINE__); \
        printf("Error: %s\n", hipGetErrorString(error)); \
        exit(1); \
    } \
} \

/********************************
* 
*   CPU Code
* 
*********************************/
// Compute image gradients using Sobel operator
void computeGradients(const unsigned char* I1, int width, int height, int stride, float* Ix, float* Iy, float* It, const unsigned char* I2) {
    for (int y = 1; y < height - 1; y++) {
        for (int x = 1; x < width - 1; x++) {
            int idx = y * stride + x;

            Ix[idx] = (I1[idx + 1] - I1[idx - 1] + I2[idx + 1] - I2[idx - 1]) / 4.0f;
            Iy[idx] = (I1[idx + stride] - I1[idx - stride] + I2[idx + stride] - I2[idx - stride]) / 4.0f;
            It[idx] = (float)(I1[idx] - I2[idx]);
        }
    }
}

// Lucas-Kanade method
void computeOpticalFlow(const float* Ix, const float* Iy, const float* It, int width, int height, int stride, float* u, float* v) {
    for (int y = WIN_SIZE / 2; y < height - WIN_SIZE / 2; y++) {
        for (int x = WIN_SIZE / 2; x < width - WIN_SIZE / 2; x++) {
            float sumIx2 = 0, sumIy2 = 0, sumIxIy = 0, sumIxIt = 0, sumIyIt = 0;

            for (int wy = -WIN_SIZE / 2; wy <= WIN_SIZE / 2; wy++) {
                for (int wx = -WIN_SIZE / 2; wx <= WIN_SIZE / 2; wx++) {
                    int idx = (y + wy) * stride + (x + wx);
                    sumIx2 += Ix[idx] * Ix[idx];
                    sumIy2 += Iy[idx] * Iy[idx];
                    sumIxIy += Ix[idx] * Iy[idx];
                    sumIxIt += Ix[idx] * It[idx];
                    sumIyIt += Iy[idx] * It[idx];
                }
            }

            float det = sumIx2 * sumIy2 - sumIxIy * sumIxIy;
            if (fabs(det) > 1e-6) {
                u[y * stride + x] = (sumIy2 * -sumIxIt - sumIxIy * -sumIyIt) / det;
                v[y * stride + x] = (sumIx2 * -sumIyIt - sumIxIy * -sumIxIt) / det;
            }
            else {
                u[y * stride + x] = 0;
                v[y * stride + x] = 0;
            }
        }
    }
}

/***************************************
*
*   GPU Code
*
****************************************/

// kernel to get the Ix, Iy and It of two images
__global__ void cudaComputeGradients(float* Ix, float* Iy, float* It, const unsigned char* I1, const unsigned char* I2, int width, int height)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int idx = i + j * width;
    if (i > 0 && i < width - 1 && j > 0 && j < height - 1)
    {
        Ix[idx] = (I1[idx + 1] - I1[idx - 1] + I2[idx + 1] - I2[idx - 1]) / 4.0f;
        Iy[idx] = (I1[idx + width] - I1[idx - width] + I2[idx + width] - I2[idx - width]) / 4.0f;
        It[idx] = (I2[idx] - I1[idx]);
    }
}


__global__ void cudaComputeOpticalFlow(const float* Ix, const float* Iy, const float* It, int width, int height, int stride, float* u, float* v)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y + (WIN_SIZE / 2);
    int x = blockIdx.x * blockDim.x + threadIdx.x + (WIN_SIZE / 2);

    if (x >= (width - WIN_SIZE / 2) || y >= (height - WIN_SIZE / 2)) {
        return;
    }

    float sumIx2 = 0, sumIy2 = 0, sumIxIy = 0, sumIxIt = 0, sumIyIt = 0;

    for (int wy = -WIN_SIZE / 2; wy <= WIN_SIZE / 2; wy++) {
        for (int wx = -WIN_SIZE / 2; wx <= WIN_SIZE / 2; wx++) {
            int idx = (y + wy) * stride + (x + wx);
            sumIx2 += Ix[idx] * Ix[idx];
            sumIy2 += Iy[idx] * Iy[idx];
            sumIxIy += Ix[idx] * Iy[idx];
            sumIxIt += Ix[idx] * It[idx];
            sumIyIt += Iy[idx] * It[idx];
        }
    }

    float det = sumIx2 * sumIy2 - sumIxIy * sumIxIy;
    if (fabs(det) > 1e-6) {
        u[y * stride + x] = (sumIy2 * -sumIxIt - sumIxIy * -sumIyIt) / det;
        v[y * stride + x] = (sumIx2 * -sumIyIt - sumIxIy * -sumIxIt) / det;
    }
    else {
        u[y * stride + x] = 0;
        v[y * stride + x] = 0;
    }
}


/*****************************************
*
*   Utils
*
******************************************/

void compare(const float* a, const float* b, int width, int height, const char* str)
{
    printf("%s\n", str);
    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            int idx = y * width + x;
            if (abs(a[idx] - b[idx]) > 1e-3)
            {
                printf("Mismatch at idx: %d, %f, %f\n", idx, a[idx], b[idx]);
            }
        }
    }
}


// Visualize optical flow as HSV
void visualizeOpticalFlow(const float* u, const float* v, int width, int height, int stride, unsigned char* output) {
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            int idx = y * stride + x;

            float magnitude = sqrt(u[idx] * u[idx] + v[idx] * v[idx]);
            float angle = (float)atan2(v[idx], u[idx]) * 180.0f / CV_PI + 180.0f; // Convert to degrees

            float normMagnitude = fmin(magnitude / 10.0f, 1.0f); // Clipping magnitude

            // Convert HSV to RGB
            float h = angle / 2.0f; // [0, 6)
            float s = 0.5f;
            float v = normMagnitude;

            output[idx * 3 + 0] = (unsigned char)(h);
            output[idx * 3 + 1] = (unsigned char)(s * 255);
            output[idx * 3 + 2] = (unsigned char)(v * 255);
        }
    }
}

void writeToFile(const float* Ix, int width, int height, const char* file_name)
{
    FILE* f = fopen(file_name, "w");
    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            fprintf(f, "%f,", Ix[i * width + j]);
        }
        fprintf(f, "\n");
    }
    fclose(f);
}

int main(int argc, char** argv) {
    if (argc != 2) {
        printf("Usage: %s <video_file>\n", argv[0]);
        return -1;
    }

    // Open video
    cv::VideoCapture cap(argv[1]);
    if (!cap.isOpened()) {
        printf("Error: Unable to open video file.\n");
        return -1;
    }

    int width = 512;
    int height = 512;
    int stride = width;
    int size = width * height * sizeof(float);

    // CPU Memory Allocation
    cv::Mat frame;
    unsigned char* temp = NULL;
    unsigned char* I1; // = (unsigned char*)malloc(height * stride);
    unsigned char* I2; // = (unsigned char*)malloc(height * stride);
    float* Ix = (float*)calloc(height * stride, sizeof(float));
    float* Iy = (float*)calloc(height * stride, sizeof(float));
    float* It = (float*)calloc(height * stride, sizeof(float));
    float* u; // = (float*)calloc(height * stride, sizeof(float));
    float* v; // = (float*)calloc(height * stride, sizeof(float));
    unsigned char* output = (unsigned char*)calloc(height * stride * 3, sizeof(unsigned char));

    // DMA Alloc
    hipHostAlloc((void**)&I1, height * width, hipHostMallocDefault);
    hipHostAlloc((void**)&I2, height * width, hipHostMallocDefault);
    hipHostAlloc((void**)&u, size, hipHostMallocDefault);
    hipHostAlloc((void**)&v, size, hipHostMallocDefault);

    // GPU Memory Allocation
    unsigned char* d_I1, * d_I2;
    float* d_Ix, * d_Iy, * d_It, * d_u, * d_v;

    hipMalloc(&d_I1, width * height);
    hipMalloc(&d_I2, width * height);
    hipMalloc(&d_Ix, size);
    hipMalloc(&d_Iy, size);
    hipMalloc(&d_It, size);
    hipMalloc(&d_u, size);
    hipMalloc(&d_v, size);

    hipMemset(&d_Ix, 0, size);
    hipMemset(&d_Iy, 0, size);
    hipMemset(&d_It, 0, size);
    hipMemset(&d_u, 0, size);
    hipMemset(&d_v, 0, size);

    // Debug 
    float* Ix_cpu = (float*)malloc(size);
    float* Iy_cpu = (float*)malloc(size);
    float* It_cpu = (float*)malloc(size);
    float* u_cpu = (float*)malloc(size);
    float* v_cpu = (float*)malloc(size);

    time_t tick = time(NULL);
    float calc_fps = 0;
    int frame_num = 0;
    while (true)
    {
        int ret = cap.read(frame);
        if (!ret) break;
        cv::cvtColor(frame, frame, cv::COLOR_BGR2GRAY);
        cv::resize(frame, frame, cv::Size(width, height));

#if DISPLAY_STREAMS
        cv::imshow("Input Frame Display", frame);
        float fps = cap.get(cv::CAP_PROP_FPS);
        int delay = static_cast<int>(1000 / fps); // Delay between frames in milliseconds

        if (cv::waitKey(delay) == 'q') {
            std::cout << "Exiting video playback" << std::endl;
            break;
        }
#endif

        frame_num++;
        temp = I1;
        I1 = I2;
        I2 = temp;

        temp = d_I1;
        d_I1 = d_I2;
        d_I2 = temp;

        memcpy(I2, frame.data, height * width * sizeof(unsigned char));
        //hipMemcpy(d_I2, frame.data, height * width * sizeof(unsigned char), hipMemcpyHostToDevice);
        hipMemcpyAsync(d_I2, I2, height * width * sizeof(unsigned char), hipMemcpyHostToDevice);

#if CPU
        computeGradients(I2, width, height, stride, Ix, Iy, It, I1);
        computeOpticalFlow(Ix, Iy, It, width, height, stride, u, v);
        //visualizeOpticalFlow(u, v, width, height, stride, output);
#endif

        dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE / N);
        dim3 numBlocks(width / threadsPerBlock.x, height / threadsPerBlock.y);      
        cudaComputeGradients<< <numBlocks, threadsPerBlock >> > (d_Ix, d_Iy, d_It, d_I1, d_I2, width, height);

#if DEBUG
        hipMemcpy(Ix_cpu, d_Ix, size, hipMemcpyDeviceToHost);
        hipMemcpy(Iy_cpu, d_Iy, size, hipMemcpyDeviceToHost);
        hipMemcpy(It_cpu, d_It, size, hipMemcpyDeviceToHost);

        if (frame_num > 1)
        {
            compare(Ix, Ix_cpu, width, height, "Ix");
            compare(Iy, Iy_cpu, width, height, "Iy");
            compare(It, It_cpu, width, height, "It");
        }

        /*writeToFile(Ix, width, height, "Ix.csv");
        writeToFile(Iy, width, height, "Iy.csv");
        writeToFile(It, width, height, "It.csv");*/
#endif

        dim3 threadsPerBlock2(BLOCK_SIZE, BLOCK_SIZE / N);
        dim3 numBlocks2(width / threadsPerBlock2.x, height / threadsPerBlock2.y);
        cudaComputeOpticalFlow << <numBlocks2, threadsPerBlock2 >> > (d_Ix, d_Iy, d_It, width, height, stride, d_u, d_v);

        hipMemcpyAsync(u_cpu, d_u, size, hipMemcpyDeviceToHost);
        hipMemcpyAsync(v_cpu, d_v, size, hipMemcpyDeviceToHost);
        //hipMemcpy(u_cpu, d_u, size, hipMemcpyDeviceToHost);
        //hipMemcpy(v_cpu, d_v, size, hipMemcpyDeviceToHost);

#if DISPLAY_STREAMS
        visualizeOpticalFlow(u_cpu, v_cpu, width, height, stride, output);
#endif 

#if DEBUG
        if (frame_num > 1)
        {
            compare(u_cpu, u, width, height, "U");
            compare(v_cpu, v, width, height, "V");
        }

       /* writeToFile(u, width, height, "U.csv");
        writeToFile(v, width, height, "V.csv");
        
        writeToFile(u_cpu, width, height, "U_gpu.csv");
        writeToFile(v_cpu, width, height, "V_gpu.csv");*/
#endif

        time_t tock = time(NULL);
        calc_fps = frame_num / (float)difftime(tock, tick);

#if DISPLAY_STREAMS
        cv::Mat opflow(height, width, CV_8UC3, output);
        cv::cvtColor(opflow, opflow, cv::COLOR_HSV2BGR);

        char text[12];
        sprintf(text, "FPS: %.2f", calc_fps);

        cv::putText(opflow, //target image
            text, //text
            cv::Point(10, opflow.rows - 10), //top-left position
            cv::FONT_HERSHEY_DUPLEX,
            1.0,
            CV_RGB(118, 185, 0), //font color
            1);

        cv::imshow("Optical Flow Display", opflow);
#endif
    }

    cout << "Calculated FPS: " << calc_fps << "\n";

    //free(I1);
    //free(I2);
    free(Ix);
    free(Iy);
    free(It);
    //free(u);
    //free(v);

    hipHostFree(I1);
    hipHostFree(I2);
    hipHostFree(u);
    hipHostFree(v);

    hipFree(d_I1);
    hipFree(d_I2);
    hipFree(d_Ix);
    hipFree(d_Iy);
    hipFree(d_It);
    hipFree(d_u);
    hipFree(d_v);

    free(Ix_cpu);
    free(Iy_cpu);
    free(It_cpu);
    free(u_cpu);
    free(v_cpu);

}